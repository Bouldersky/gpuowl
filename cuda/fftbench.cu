#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <initializer_list>
#include <algorithm>
#include <vector>

#include "timeutil.h"

#define CHECK(what) assert(what == hipSuccess)

bool isGood(int n) {
  int a = 0, b = 0, c = 0, d = 0;
  while (n % 2 == 0) { n /= 2; ++a; }
  while (n % 3 == 0) { n /= 3; ++b; }
  while (n % 5 == 0) { n /= 5; ++c; }
  while (n % 7 == 0) { n /= 7; ++d; }
  bool good = n == 1;
  if (good) { printf("[%2d %2d %2d %2d] ", a, b, c, d); }
  return good;
}

int main() {
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  
  Timer timer;
  int N = 16 * 1024 * 1024;
  double2 *data = new double2[N]();
  /*
  for (int i = 0; i < N; ++i) {
    data[i].x = i;
    data[i].y = i / 2;
  }
  printf("init data %d\n", timer.deltaMillis());
  */
  
  double2 *buf1;
  // double2 *buf2;
  CHECK(hipMalloc((void **)&buf1, N * sizeof(double2)));
  CHECK(hipMemcpy(buf1, data, N * sizeof(double2), hipMemcpyHostToDevice));
  
  // CHECK(hipMalloc((void **)&buf2, N * sizeof(double2)));
  // CHECK(hipMemcpy(buf2, data, N * sizeof(double2), hipMemcpyHostToDevice));
  // printf("copy %d\n", timer.deltaMillis());

  std::vector<std::pair<float, int>> v;
  for (int k = 2048*1024; k <= 11 * 1024*1024; k += 1024) {
    if (!isGood(k)) { continue; }
    int size = k;
    hipfftHandle plan;
    CHECK(hipfftPlan1d(&plan, size, HIPFFT_Z2Z, 1));
    size_t planSize;
    CHECK(hipfftGetSize(plan, &planSize));
    timer.deltaMillis();
    int reps = 400;
    for (int i = 0; i < reps; ++i) { CHECK(hipfftExecZ2Z(plan, buf1, buf1, HIPFFT_FORWARD)); }
    CHECK(hipDeviceSynchronize());
    int t = timer.deltaMillis();
    float tt = t / float(reps);
    printf("%5dK %2.2fms %.2f (%d MB)\n", size / 1024, tt, t * 1024 / float(k), int(planSize / (1024 * 1024)));
    while (!v.empty() && v.back().first > tt) { v.pop_back(); }
    v.push_back(std::make_pair(tt, k));
    CHECK(hipfftDestroy(plan));
  }

  printf("\n----\n");
  for (auto x : v) {
    isGood(x.second);
    printf("%.1f %.2f\n", x.second / float(1024), x.first);
  }
  CHECK(hipFree(buf1));
  // CHECK(hipFree(buf2));
}
