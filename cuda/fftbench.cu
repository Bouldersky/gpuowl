#include "hip/hip_runtime.h"
#include <hipfft/hipfft.h>

#include <stdio.h>
#include <assert.h>
#include <math.h>
#include <initializer_list>
#include <algorithm>
#include <vector>

#include "timeutil.h"

#define CHECK(what) assert(what == hipSuccess)

bool isGood(int n) {
  int a = 0, b = 0, c = 0, d = 0;
  while (n % 2 == 0) { n /= 2; ++a; }
  while (n % 3 == 0) { n /= 3; ++b; }
  while (n % 5 == 0) { n /= 5; ++c; }
  while (n % 7 == 0) { n /= 7; ++d; }
  bool good = n == 1;
  if (good) { printf("[%2d %2d %2d %2d] ", a, b, c, d); }
  return good;
}

int main(int argc, char **argv) {
  hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
  
  Timer timer;
  int N = 8 * 1024 * 1024;
  double *data = new double[N]();  
  double *buf1;
  CHECK(hipMalloc((void **)&buf1, (N + 1) * sizeof(double)));
  CHECK(hipMemcpy(buf1, data, N * sizeof(double), hipMemcpyHostToDevice));

  double *buf2;
  CHECK(hipMalloc((void **)&buf2, (N + 1) * sizeof(double)));
  CHECK(hipMemcpy(buf2, data, N * sizeof(double), hipMemcpyHostToDevice));

  std::vector<std::pair<float, int>> v;
  for (int k = 4*1024*1024; k <= N; k += 2048) {
    if (!isGood(k)) { continue; }
    int size = k;
    hipfftHandle plan1, plan2;
    CHECK(hipfftPlan1d(&plan1, size, HIPFFT_D2Z, 1));
    CHECK(hipfftPlan1d(&plan2, size, HIPFFT_Z2D, 1));
    size_t planSize1, planSize2;
    CHECK(hipfftGetSize(plan1, &planSize1));
    CHECK(hipfftGetSize(plan2, &planSize2));
    
    timer.deltaMillis();
    int reps = 400;
    
    for (int i = 0; i < reps / 2; ++i) {
      CHECK(hipfftExecD2Z(plan1, buf1, (double2 *) buf2));
      CHECK(hipfftExecZ2D(plan2, (double2 *) buf2, buf1));
    }
    CHECK(hipDeviceSynchronize());
    float t1 = timer.deltaMillis() / float(reps);
    printf("%5dK %2.2fms (%d MB)\n", size / 1024, t1, int(planSize1 / (1024 * 1024)));

    /*
    for (int i = 0; i < reps; ++i) { CHECK(hipfftExecZ2D(plan2, (double2 *) buf1, buf1)); }
    CHECK(hipDeviceSynchronize());
    float t2 = timer.deltaMillis() / float(reps);
    printf("%5dK %2.2fms (%d MB)\n", size / 1024, t2, int(planSize2 / (1024 * 1024)));
    */
    
    float tt = t1;
    while (!v.empty() && v.back().first > tt) { v.pop_back(); }
    v.push_back(std::make_pair(tt, k));
    
    CHECK(hipfftDestroy(plan1));
    CHECK(hipfftDestroy(plan2));
  }

  printf("\n----\n");
  for (auto x : v) {
    isGood(x.second);
    printf("%.1f %.2f\n", x.second / float(1024), x.first);
  }
  CHECK(hipFree(buf1));
  CHECK(hipFree(buf2));
}
